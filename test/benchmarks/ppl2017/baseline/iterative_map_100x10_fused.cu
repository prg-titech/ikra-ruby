#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#define GRID_DIM 58594
#define BLOCK_DIM 1024

using namespace std;

__global__ void kernel_new(int *data) {
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    // int indices[] = {idx_0, idx_1, idx_2, idx_3};

    data[_tid_] = idx_2 % 133777;
}

/*
    RUBY:

    y = y.pmap(with_index: true) do |i, indices|
        (i + indices[2]) % 13377
    end

    y = y.pmap(with_index: true) do |i, indices|
        (i + indices[1]) % 13377
    end

    y = y.pmap(with_index: true) do |i, indices|
        (i + indices[3]) % 1337
    end

    y = y.pmap(with_index: true) do |i, indices|
        (i + indices[0]) % 13377
    end

    y = y.pmap(with_index: true) do |i, indices|
        (i + indices[1]) % 1377
    end
*/

__global__ void kernel_1(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (((((((((data[_tid_] + idx_2) % 13377) + idx_1) % 13377) + idx_3) % 1337) + idx_0) % 13377) + idx_1) % 1377;
}

int main()
{
    long time_kernel = 0;
    long time_alloc = 0;
    long time_free = 0;
    long time_transfer = 0;
    long time_setup = 0;

    auto start_time = chrono::high_resolution_clock::now();
    auto end_time = chrono::high_resolution_clock::now();

    // Init
    start_time = chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
    hipFree(0);
    end_time = chrono::high_resolution_clock::now();;
    time_setup = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    auto start_entire = chrono::high_resolution_clock::now();
    
    // Measure kernel invocation
    printf("START\n");
    start_time = chrono::high_resolution_clock::now();
    int * data;
    hipMalloc(&data, (sizeof(int) * 60000000));
    hipDeviceSynchronize();

    end_time = chrono::high_resolution_clock::now();
    time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

 
    start_time = chrono::high_resolution_clock::now();
    kernel_new<<<GRID_DIM, BLOCK_DIM>>>(data);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    for (int r = 0; r < 100; r++)
    {
        // ITERATION
        start_time = chrono::high_resolution_clock::now();
        int * new_data;
        hipMalloc(&new_data, (sizeof(int) * 60000000));
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        kernel_1<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        hipFree(data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        data = new_data;
    }

    hipDeviceSynchronize();

    // Copy back
    start_time = chrono::high_resolution_clock::now();
    int * tmp_result = (int *) malloc(sizeof(int) * 60000000);
    hipMemcpy(tmp_result, data, sizeof(int) * 60000000, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_transfer += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();


    end_time = chrono::high_resolution_clock::now();
    int time_entire = chrono::duration_cast<chrono::microseconds>(end_time - start_entire).count();

    printf("setup: %f\n", time_setup / 1000.0f);
    printf("alloc: %f\n", time_alloc / 1000.0);
    printf("kernel: %f\n", time_kernel / 1000.0);
    printf("transfer: %f\n", time_transfer / 1000.0f);
    printf("free: %f\n", time_free / 1000.f);
    printf("rest: %f\n", (time_entire - time_alloc - time_kernel - time_transfer - time_free) / 1000.0f);

    printf("END\n");  
}
