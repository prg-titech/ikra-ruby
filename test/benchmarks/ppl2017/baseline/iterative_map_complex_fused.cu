#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#define GRID_DIM 58594
#define BLOCK_DIM 1024

using namespace std;

/*

        base = Array.pnew(dimensions: DIMS) do |indices|
            (indices[2]) % 133777
        end

        return Ikra::Symbolic.host_section(base) do |x|
            y = x
            old_data = x.__call__.to_command

            for r in 0...200
                if r % 2 == 0
                    if r % 3 == 0
                        y = y.pmap(with_index: true) do |i, indices|
                            (i + indices[3]) % 77689
                        end
                    else
                        y = y.pmap(with_index: true) do |i, indices|
                            (i + indices[0]) % 11799
                        end
                    end
                else
                    y = y.pmap(with_index: true) do |i, indices|
                        (i + indices[2]) % 1337
                    end

                    y = y.pmap(with_index: true) do |i, indices|
                        (i + indices[2]) % 8888888
                    end
                end

                y = y.pmap(with_index: true) do |i, indices|
                    (i + indices[2]) % 6678
                end

                old_data.free_memory
                old_data = y
            end

            y
        end

*/

__global__ void kernel_new(int *data) {
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    // int indices[] = {idx_0, idx_1, idx_2, idx_3};

    data[_tid_] = idx_2 % 133777;
}

__global__ void kernel_1(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (data[_tid_] + idx_3) % 77689;
}

__global__ void kernel_2(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (data[_tid_] + idx_0) % 11799;
}

__global__ void kernel_3(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (data[_tid_] + idx_2) % 1337;
}

__global__ void kernel_4(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (data[_tid_] + idx_2) % 8888888;
}

__global__ void kernel_5(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 60000000) return;

    int idx_0 = _tid_ / (12*500*500);
    int idx_1 = (_tid_ / (12*500)) % 500;
    int idx_2 = (_tid_ / 12) % 500;
    int idx_3 = (_tid_ / 1) % 12;

    new_data[_tid_] = (data[_tid_] + idx_2) % 6678;
}

int main()
{
    long time_kernel = 0;
    long time_alloc = 0;
    long time_free = 0;
    long time_transfer = 0;

    auto start_time = chrono::high_resolution_clock::now();
    auto end_time = chrono::high_resolution_clock::now();

    // Init
    start_time = chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
    hipFree(0);
    end_time = chrono::high_resolution_clock::now();;
    long time_setup = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    auto start_entire = chrono::high_resolution_clock::now();

    printf("START\n");
    start_time = chrono::high_resolution_clock::now();
    int * data;
    hipMalloc(&data, (sizeof(int) * 60000000));
    hipDeviceSynchronize();

    end_time = chrono::high_resolution_clock::now();
    time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

 
    start_time = chrono::high_resolution_clock::now();
    kernel_new<<<GRID_DIM, BLOCK_DIM>>>(data);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    for (int r = 0; r < 200; r++)
    {
        if (r % 2 == 0) {
            if (r % 3 == 0) {
                // KERNEL LAUNCH
                start_time = chrono::high_resolution_clock::now();
                int * new_data;
                hipMalloc(&new_data, (sizeof(int) * 60000000));
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                start_time = chrono::high_resolution_clock::now();
                kernel_1<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                start_time = chrono::high_resolution_clock::now();
                hipFree(data);
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                data = new_data;
            }
            else {
                // KERNEL LAUNCH
                start_time = chrono::high_resolution_clock::now();
                int * new_data;
                hipMalloc(&new_data, (sizeof(int) * 60000000));
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                start_time = chrono::high_resolution_clock::now();
                kernel_2<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                start_time = chrono::high_resolution_clock::now();
                hipFree(data);
                hipDeviceSynchronize();
                end_time = chrono::high_resolution_clock::now();
                time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

                data = new_data;
            }
        } else {
            // KERNEL LAUNCH
            start_time = chrono::high_resolution_clock::now();
            int * new_data;
            hipMalloc(&new_data, (sizeof(int) * 60000000));
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            start_time = chrono::high_resolution_clock::now();
            kernel_3<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            start_time = chrono::high_resolution_clock::now();
            hipFree(data);
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            data = new_data;

            // KERNEL LAUNCH
            start_time = chrono::high_resolution_clock::now();
            hipMalloc(&new_data, (sizeof(int) * 60000000));
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            start_time = chrono::high_resolution_clock::now();
            kernel_4<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            start_time = chrono::high_resolution_clock::now();
            hipFree(data);
            hipDeviceSynchronize();
            end_time = chrono::high_resolution_clock::now();
            time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

            data = new_data;
        }

        // KERNEL LAUNCH
        start_time = chrono::high_resolution_clock::now();
        int * new_data;
        hipMalloc(&new_data, (sizeof(int) * 60000000));
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        kernel_5<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        hipFree(data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        data = new_data;
    }

    hipDeviceSynchronize();

    // Copy back
    start_time = chrono::high_resolution_clock::now();
    int * tmp_result = (int *) malloc(sizeof(int) * 60000000);
    hipMemcpy(tmp_result, data, sizeof(int) * 60000000, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_transfer += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();


    end_time = chrono::high_resolution_clock::now();
    int time_entire = chrono::duration_cast<chrono::microseconds>(end_time - start_entire).count();

    printf("alloc: %f\n", time_alloc / 1000.0);
    printf("kernel: %f\n", time_kernel / 1000.0);
    printf("transfer: %f\n", time_transfer / 1000.0f);
    printf("free: %f\n", time_free / 1000.f);
    printf("rest: %f\n", (time_entire - time_alloc - time_kernel - time_transfer - time_free) / 1000.0f);

    printf("END\n");  
}
