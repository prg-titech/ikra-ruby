#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
struct indexed_struct_4_lt_int_int_int_int_gt_t
{
    int field_0;
int field_1;
int field_2;
int field_3;
};

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct array_command_1 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_1(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_2 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_1 *input_0;
    __host__ __device__ array_command_2(int *result = NULL, array_command_1 *input_0 = NULL) : result(result), input_0(input_0) { }
};
struct array_command_4 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_4(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_3 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_2 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_3(int *result = NULL, array_command_2 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_5 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_3 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_5(int *result = NULL, array_command_3 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_7 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_5 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_7(int *result = NULL, array_command_5 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_9 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_7 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_9(int *result = NULL, array_command_7 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_11 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_9 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_11(int *result = NULL, array_command_9 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_13 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_11 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_13(int *result = NULL, array_command_11 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_15 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_13 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_15(int *result = NULL, array_command_13 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_17 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_15 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_17(int *result = NULL, array_command_15 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_19 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_17 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_19(int *result = NULL, array_command_17 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct array_command_21 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_19 *input_0;
    array_command_4 *input_1;
    __host__ __device__ array_command_21(int *result = NULL, array_command_19 *input_0 = NULL, array_command_4 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct environment_struct
{
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((indices.field_0 + indices.field_1)) % ((((indices.field_3 + ({ int _temp_var_1 = ((indices.field_1 % 4));
        (_temp_var_1 == 0 ? indices.field_0 : (_temp_var_1 == 1 ? indices.field_1 : (_temp_var_1 == 2 ? indices.field_2 : (_temp_var_1 == 3 ? indices.field_3 : NULL)))); }))) + 7)));
    }
}

#endif


__global__ void kernel_21(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
}



// TODO: There should be a better to check if _block_k_3_ is already defined
#ifndef _block_k_3__func
#define _block_k_3__func
__device__ int _block_k_3_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_2 = ((({ int _temp_var_3 = ((({ int _temp_var_4 = ((values[2] % 4));
        (_temp_var_4 == 0 ? indices.field_0 : (_temp_var_4 == 1 ? indices.field_1 : (_temp_var_4 == 2 ? indices.field_2 : (_temp_var_4 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_3 == 0 ? indices.field_0 : (_temp_var_3 == 1 ? indices.field_1 : (_temp_var_3 == 2 ? indices.field_2 : (_temp_var_3 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_2 == 0 ? indices.field_0 : (_temp_var_2 == 1 ? indices.field_1 : (_temp_var_2 == 2 ? indices.field_2 : (_temp_var_2 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_19(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_22)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_23;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_23 = _block_k_3_(_env_, _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_23 = 37;
    }
        
        _result_[_tid_] = temp_stencil_23;
    }
}



// TODO: There should be a better to check if _block_k_5_ is already defined
#ifndef _block_k_5__func
#define _block_k_5__func
__device__ int _block_k_5_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_5 = ((({ int _temp_var_6 = ((({ int _temp_var_7 = ((values[2] % 4));
        (_temp_var_7 == 0 ? indices.field_0 : (_temp_var_7 == 1 ? indices.field_1 : (_temp_var_7 == 2 ? indices.field_2 : (_temp_var_7 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_6 == 0 ? indices.field_0 : (_temp_var_6 == 1 ? indices.field_1 : (_temp_var_6 == 2 ? indices.field_2 : (_temp_var_6 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_5 == 0 ? indices.field_0 : (_temp_var_5 == 1 ? indices.field_1 : (_temp_var_5 == 2 ? indices.field_2 : (_temp_var_5 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_17(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_20)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_24;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_24 = _block_k_5_(_env_, _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_24 = 37;
    }
        
        _result_[_tid_] = temp_stencil_24;
    }
}



// TODO: There should be a better to check if _block_k_7_ is already defined
#ifndef _block_k_7__func
#define _block_k_7__func
__device__ int _block_k_7_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_8 = ((({ int _temp_var_9 = ((({ int _temp_var_10 = ((values[2] % 4));
        (_temp_var_10 == 0 ? indices.field_0 : (_temp_var_10 == 1 ? indices.field_1 : (_temp_var_10 == 2 ? indices.field_2 : (_temp_var_10 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_9 == 0 ? indices.field_0 : (_temp_var_9 == 1 ? indices.field_1 : (_temp_var_9 == 2 ? indices.field_2 : (_temp_var_9 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_8 == 0 ? indices.field_0 : (_temp_var_8 == 1 ? indices.field_1 : (_temp_var_8 == 2 ? indices.field_2 : (_temp_var_8 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_15(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_18)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_25;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_25 = _block_k_7_(_env_, _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_25 = 37;
    }
        
        _result_[_tid_] = temp_stencil_25;
    }
}



// TODO: There should be a better to check if _block_k_9_ is already defined
#ifndef _block_k_9__func
#define _block_k_9__func
__device__ int _block_k_9_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_11 = ((({ int _temp_var_12 = ((({ int _temp_var_13 = ((values[2] % 4));
        (_temp_var_13 == 0 ? indices.field_0 : (_temp_var_13 == 1 ? indices.field_1 : (_temp_var_13 == 2 ? indices.field_2 : (_temp_var_13 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_12 == 0 ? indices.field_0 : (_temp_var_12 == 1 ? indices.field_1 : (_temp_var_12 == 2 ? indices.field_2 : (_temp_var_12 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_11 == 0 ? indices.field_0 : (_temp_var_11 == 1 ? indices.field_1 : (_temp_var_11 == 2 ? indices.field_2 : (_temp_var_11 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_13(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_16)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_26;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_26 = _block_k_9_(_env_, _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_26 = 37;
    }
        
        _result_[_tid_] = temp_stencil_26;
    }
}



// TODO: There should be a better to check if _block_k_11_ is already defined
#ifndef _block_k_11__func
#define _block_k_11__func
__device__ int _block_k_11_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_14 = ((({ int _temp_var_15 = ((({ int _temp_var_16 = ((values[2] % 4));
        (_temp_var_16 == 0 ? indices.field_0 : (_temp_var_16 == 1 ? indices.field_1 : (_temp_var_16 == 2 ? indices.field_2 : (_temp_var_16 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_15 == 0 ? indices.field_0 : (_temp_var_15 == 1 ? indices.field_1 : (_temp_var_15 == 2 ? indices.field_2 : (_temp_var_15 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_14 == 0 ? indices.field_0 : (_temp_var_14 == 1 ? indices.field_1 : (_temp_var_14 == 2 ? indices.field_2 : (_temp_var_14 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_11(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_14)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_27;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_27 = _block_k_11_(_env_, _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_27 = 37;
    }
        
        _result_[_tid_] = temp_stencil_27;
    }
}



// TODO: There should be a better to check if _block_k_13_ is already defined
#ifndef _block_k_13__func
#define _block_k_13__func
__device__ int _block_k_13_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_17 = ((({ int _temp_var_18 = ((({ int _temp_var_19 = ((values[2] % 4));
        (_temp_var_19 == 0 ? indices.field_0 : (_temp_var_19 == 1 ? indices.field_1 : (_temp_var_19 == 2 ? indices.field_2 : (_temp_var_19 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_18 == 0 ? indices.field_0 : (_temp_var_18 == 1 ? indices.field_1 : (_temp_var_18 == 2 ? indices.field_2 : (_temp_var_18 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_17 == 0 ? indices.field_0 : (_temp_var_17 == 1 ? indices.field_1 : (_temp_var_17 == 2 ? indices.field_2 : (_temp_var_17 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_9(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_12)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_28;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_28 = _block_k_13_(_env_, _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_28 = 37;
    }
        
        _result_[_tid_] = temp_stencil_28;
    }
}



// TODO: There should be a better to check if _block_k_15_ is already defined
#ifndef _block_k_15__func
#define _block_k_15__func
__device__ int _block_k_15_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_20 = ((({ int _temp_var_21 = ((({ int _temp_var_22 = ((values[2] % 4));
        (_temp_var_22 == 0 ? indices.field_0 : (_temp_var_22 == 1 ? indices.field_1 : (_temp_var_22 == 2 ? indices.field_2 : (_temp_var_22 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_21 == 0 ? indices.field_0 : (_temp_var_21 == 1 ? indices.field_1 : (_temp_var_21 == 2 ? indices.field_2 : (_temp_var_21 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_20 == 0 ? indices.field_0 : (_temp_var_20 == 1 ? indices.field_1 : (_temp_var_20 == 2 ? indices.field_2 : (_temp_var_20 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_7(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_10)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_29;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_29 = _block_k_15_(_env_, _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_29 = 37;
    }
        
        _result_[_tid_] = temp_stencil_29;
    }
}



// TODO: There should be a better to check if _block_k_17_ is already defined
#ifndef _block_k_17__func
#define _block_k_17__func
__device__ int _block_k_17_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_23 = ((({ int _temp_var_24 = ((({ int _temp_var_25 = ((values[2] % 4));
        (_temp_var_25 == 0 ? indices.field_0 : (_temp_var_25 == 1 ? indices.field_1 : (_temp_var_25 == 2 ? indices.field_2 : (_temp_var_25 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_24 == 0 ? indices.field_0 : (_temp_var_24 == 1 ? indices.field_1 : (_temp_var_24 == 2 ? indices.field_2 : (_temp_var_24 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_23 == 0 ? indices.field_0 : (_temp_var_23 == 1 ? indices.field_1 : (_temp_var_23 == 2 ? indices.field_2 : (_temp_var_23 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_5(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_8)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_30;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_30 = _block_k_17_(_env_, _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_30 = 37;
    }
        
        _result_[_tid_] = temp_stencil_30;
    }
}



// TODO: There should be a better to check if _block_k_19_ is already defined
#ifndef _block_k_19__func
#define _block_k_19__func
__device__ int _block_k_19_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_26 = ((({ int _temp_var_27 = ((({ int _temp_var_28 = ((values[2] % 4));
        (_temp_var_28 == 0 ? indices.field_0 : (_temp_var_28 == 1 ? indices.field_1 : (_temp_var_28 == 2 ? indices.field_2 : (_temp_var_28 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_27 == 0 ? indices.field_0 : (_temp_var_27 == 1 ? indices.field_1 : (_temp_var_27 == 2 ? indices.field_2 : (_temp_var_27 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_26 == 0 ? indices.field_0 : (_temp_var_26 == 1 ? indices.field_1 : (_temp_var_26 == 2 ? indices.field_2 : (_temp_var_26 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_3(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_6)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_31;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_31 = _block_k_19_(_env_, _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_31 = 37;
    }
        
        _result_[_tid_] = temp_stencil_31;
    }
}



// TODO: There should be a better to check if _block_k_21_ is already defined
#ifndef _block_k_21__func
#define _block_k_21__func
__device__ int _block_k_21_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_29 = ((({ int _temp_var_30 = ((({ int _temp_var_31 = ((values[2] % 4));
        (_temp_var_31 == 0 ? indices.field_0 : (_temp_var_31 == 1 ? indices.field_1 : (_temp_var_31 == 2 ? indices.field_2 : (_temp_var_31 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_30 == 0 ? indices.field_0 : (_temp_var_30 == 1 ? indices.field_1 : (_temp_var_30 == 2 ? indices.field_2 : (_temp_var_30 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_29 == 0 ? indices.field_0 : (_temp_var_29 == 1 ? indices.field_1 : (_temp_var_29 == 2 ? indices.field_2 : (_temp_var_29 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_1(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_4)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_32;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 3000000;
int temp_stencil_dim_1 = (_tid_ / 6000) % 500;
int temp_stencil_dim_2 = (_tid_ / 12) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 12;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 12)
    {
        // All value indices within bounds
        
        temp_stencil_32 = _block_k_21_(_env_, _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 0) * 3000000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + 0) * 6000 + (temp_stencil_dim_0 + 1) * 3000000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 12 + (temp_stencil_dim_1 + -1) * 6000 + (temp_stencil_dim_0 + -1) * 3000000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 3000000, (_tid_ / 6000) % 500, (_tid_ / 12) % 500, (_tid_ / 1) % 12}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_32 = 37;
    }
        
        _result_[_tid_] = temp_stencil_32;
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return variable_size_array_t::error_return_value;\
}

variable_size_array_t _host_section__(environment_t *host_env, environment_t *dev_env, result_t *program_result)
{
    array_command_2 * base = new array_command_2();
    array_command_21 * _ssa_var_base_10;
    array_command_19 * _ssa_var_base_9;
    array_command_17 * _ssa_var_base_8;
    array_command_15 * _ssa_var_base_7;
    array_command_13 * _ssa_var_base_6;
    array_command_11 * _ssa_var_base_5;
    array_command_9 * _ssa_var_base_4;
    array_command_7 * _ssa_var_base_3;
    array_command_5 * _ssa_var_base_2;
    array_command_3 * _ssa_var_base_1;
    {
        _ssa_var_base_1 = new array_command_3(NULL, base);
        _ssa_var_base_2 = new array_command_5(NULL, _ssa_var_base_1);
        _ssa_var_base_3 = new array_command_7(NULL, _ssa_var_base_2);
        _ssa_var_base_4 = new array_command_9(NULL, _ssa_var_base_3);
        _ssa_var_base_5 = new array_command_11(NULL, _ssa_var_base_4);
        _ssa_var_base_6 = new array_command_13(NULL, _ssa_var_base_5);
        _ssa_var_base_7 = new array_command_15(NULL, _ssa_var_base_6);
        _ssa_var_base_8 = new array_command_17(NULL, _ssa_var_base_7);
        _ssa_var_base_9 = new array_command_19(NULL, _ssa_var_base_8);
        _ssa_var_base_10 = new array_command_21(NULL, _ssa_var_base_9);
        return ({
            // [Ikra::Symbolic::ArrayStencilCommand, size = 60000000]: [SendNode: [LVarReadNode: _ssa_var_base_9].pstencil([ArrayNode: [[ArrayNode: [<-1>, <0>, <0>, <0>]], [ArrayNode: [<0>, <0>, <0>, <0>]], [ArrayNode: [<1>, <0>, <0>, <0>]], [ArrayNode: [<-1>, <-1>, <0>, <0>]]]]; <37>; [HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_21 * cmd = _ssa_var_base_10;
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_22;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_22, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_22);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_21<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_22);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            int * _kernel_result_20;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_20, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_20);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_19<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_20, _kernel_result_22);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_22));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_22),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_18;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_18, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_18);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_17<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_18, _kernel_result_20);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_20));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_20),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_16;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_16, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_16);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_15<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_16, _kernel_result_18);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_18));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_18),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_14;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_14, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_14);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_13<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_14, _kernel_result_16);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_16));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_16),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_12;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_12, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_12);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_11<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_12, _kernel_result_14);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_14));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_14),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_10;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_10, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_10);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_9<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_10, _kernel_result_12);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_12));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_12),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_8;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_8, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_8);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_7<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_8, _kernel_result_10);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_10));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_10),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_6;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_6, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_6);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_5<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_6, _kernel_result_8);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_8));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_8),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_4;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_4, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_4);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_3<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_4, _kernel_result_6);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_6));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_6),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
            timeStartMeasure();
            int * _kernel_result_2;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_2, (sizeof(int) * 60000000)));
            program_result->device_allocations->push_back(_kernel_result_2);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_1<<<58594, 1024>>>(dev_env, 60000000, _kernel_result_2, _kernel_result_4);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);    timeStartMeasure();
            checkErrorReturn(program_result, hipFree(_kernel_result_4));
            program_result->device_allocations->erase(
                std::remove(
                    program_result->device_allocations->begin(),
                    program_result->device_allocations->end(),
                    _kernel_result_4),
                program_result->device_allocations->end());
            timeReportMeasure(program_result, free_memory);
        
                cmd->result = _kernel_result_2;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 60000000);
        });
    }
}

#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    


    /* Copy back memory and set pointer of result */
    program_result->result = ({
    variable_size_array_t device_array = _host_section__(host_env, dev_env, program_result);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
    timeStartMeasure();

    for (
        auto device_ptr = program_result->device_allocations->begin(); 
        device_ptr < program_result->device_allocations->end(); 
        device_ptr++)
    {
        checkErrorReturn(program_result, hipFree(*device_ptr));
    }

    delete program_result->device_allocations;

    timeReportMeasure(program_result, free_memory);

    return program_result;
}
