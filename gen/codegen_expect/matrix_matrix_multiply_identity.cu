#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */


struct environment_struct
{
    int l2_size;
    int * l2_a;
    int * l2_b;
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, int index)
{
    
    
    int i;
    int result;
    int y;
    int x;
    int * lex_b = _env_->l2_b;
    int * lex_a = _env_->l2_a;
    int lex_size = _env_->l2_size;
    {
        x = ((index % lex_size));
        y = ((index / lex_size));
        result = 0;
        for (i = 0; i <= (lex_size - 1); i++)
        {
            result = ((result + ((lex_a[((((y * lex_size)) + i))] * lex_b[((((i * lex_size)) + x))]))));
        }
        i--;
        return result;
    }
}

#endif


__global__ void kernel_38(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, _tid_);
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
    
    void * temp_ptr_l2_a = host_env->l2_a;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l2_a, 400));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(host_env->l2_a, temp_ptr_l2_a, 400, hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);

    void * temp_ptr_l2_b = host_env->l2_b;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l2_b, 400));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(host_env->l2_b, temp_ptr_l2_b, 400, hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    

    /* Launch all kernels */
        timeStartMeasure();
    int * _kernel_result_39;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_39, (sizeof(int) * 100)));
    program_result->device_allocations->push_back(_kernel_result_39);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_38<<<1, 100>>>(dev_env, 100, _kernel_result_39);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);

    /* Copy over result to the host */
    program_result->result = ({
    variable_size_array_t device_array = variable_size_array_t((void *) _kernel_result_39, 100);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
        timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_39));
    program_result->device_allocations->erase(
        std::remove(
            program_result->device_allocations->begin(),
            program_result->device_allocations->end(),
            _kernel_result_39),
        program_result->device_allocations->end());
    timeReportMeasure(program_result, free_memory);


    delete program_result->device_allocations;
    
    return program_result;
}
